#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel for image resizing
__global__ void resizeImage(const unsigned char* src, unsigned char* dst, int srcWidth, int srcHeight, int dstWidth, int dstHeight)
{
    int dstX = blockIdx.x * blockDim.x + threadIdx.x;
    int dstY = blockIdx.y * blockDim.y + threadIdx.y;

    if (dstX < dstWidth && dstY < dstHeight)
    {
        float srcX = static_cast<float>(dstX) / dstWidth * srcWidth;
        float srcY = static_cast<float>(dstY) / dstHeight * srcHeight;

        int srcX1 = static_cast<int>(srcX);
        int srcY1 = static_cast<int>(srcY);

        int srcIndex = srcY1 * srcWidth + srcX1;
        int dstIndex = dstY * dstWidth + dstX;

        dst[dstIndex] = src[srcIndex];
    }
}

int main()
{
    // Load PGM image
    const char* filename = "input2.pgm";
    FILE* file = fopen(filename, "rb");
    if (!file)
    {
        printf("Failed to open file: %s\n", filename);
        return 1;
    }

    char header[3];
    int width, height, maxVal;
    fread(header, 1, 3, file);
    fscanf(file, "%d %d %d", &width, &height, &maxVal);

    int imageSize = width * height;
    unsigned char* h_srcImage = new unsigned char[imageSize];
    fread(h_srcImage, 1, imageSize, file);
    fclose(file);

    // Allocate memory on GPU
    unsigned char* d_srcImage, * d_dstImage;
    hipMalloc(&d_srcImage, imageSize * sizeof(unsigned char));
    hipMalloc(&d_dstImage, imageSize * sizeof(unsigned char));

    // Copy input image from host to device
    hipMemcpy(d_srcImage, h_srcImage, imageSize * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Define grid and block sizes for CUDA kernel
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // Call CUDA kernel for image resizing
    resizeImage<<<gridSize, blockSize>>>(d_srcImage, d_dstImage, width, height, width / 2, height / 2);

    // Copy output image from device to host
    unsigned char* h_dstImage = new unsigned char[imageSize / 4];
    hipMemcpy(h_dstImage, d_dstImage, imageSize / 4 * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Save resized image to file
    const char* outFilename = "output2.pgm";
    FILE* outFile = fopen(outFilename, "wb");
    if (!outFile)
    {
        printf("Failed to create output file: %s\n", outFilename);
        return 1;
    }

    fprintf(outFile, "P5\n%d %d\n%d\n", width / 2, height / 2, maxVal);
    fwrite(h_dstImage, 1, imageSize / 4, outFile);
    fclose(outFile);

    // Clean up
    delete[] h_srcImage;
    delete[] h_dstImage;
    hipFree(d_srcImage);
    hipFree(d_dstImage);

    return 0;
}
